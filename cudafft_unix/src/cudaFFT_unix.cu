#include "hip/hip_runtime.h"

#include ""
//#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <pthread.h>
#include <unistd.h>
#define HAVE_REMOTE// for pcap
#include <pcap.h> /* if this gives you an error try pcap/pcap.h */
#define HR2D_PK//
#define FRAME_LEN 2048
#define OUTPUT_FRAME_SIZE FRAME_LEN*2+FRAME_HEADER_SIZE
#define FFT_SIZE_MAX 256
#define BANG_KHONG 0
int mFFTSize = 32;
#define FFT_STEP (mFFTSize / 4)

#define MAX_IREC 2400
//file mapping
#define FRAME_HEADER_SIZE 34
using namespace std;
bool isPaused = false;
// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "udpclient.h"
// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

hipfftComplex *ramSignalTL;
hipfftComplex ramSignalNen[MAX_IREC][FRAME_LEN];
hipfftComplex ramImage[FRAME_LEN];
__global__ void complexMulKernel(hipfftComplex *res, const hipfftComplex *v1, const hipfftComplex *v2)
{
	int i = threadIdx.x;
	res[i].x = (v1[i].x * v2[i].x + v1[i].y * (v2[i].y));
	res[i].y = (v1[i].x * (-v2[i].y) + v1[i].y * v2[i].x);
}
class coreFFT
{
public:

	bool isActive;
	hipfftHandle planTL;
	//hipfftHandle planNenTH;
	//hipfftHandle planImageFFT;
	hipfftComplex *dSignalTL;
	//hipfftComplex *dSignalNenRes;
	//hipfftComplex *dSignalNen;
	//hipfftComplex *dImageNen;
	int mMemSizeTL;
	//int mMemSizeImage;
	int mTichLuySize;//16
	int mFrameLen;
	coreFFT(int frameLen, int ntichluy)
	{
		isActive = false;
		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			isActive = false;
			return;
		}
		else
		{
			printf("\ncudaSetDevice on ");
			printf("\ncuda fft size:%d", ntichluy);
			printf("\nFFT ratio:1/%d", FFT_STEP);
		}
		mFrameLen = frameLen;
		mTichLuySize = ntichluy;
		mMemSizeTL = sizeof(hipfftComplex)* mTichLuySize*frameLen;
		if (hipfftPlan1d(&planTL, mTichLuySize, HIPFFT_C2C, frameLen) != HIPFFT_SUCCESS)
		{
			printf("\nFFT planTL failed to init");
			isActive = false;
			return;
		}
		ramSignalTL = new hipfftComplex[FRAME_LEN*mTichLuySize];
		// Allocate device memory for signal tich luy
		hipMalloc((void **)&dSignalTL, mMemSizeTL);
		isActive = true;
	}
	void exeFFTTL(hipfftComplex *h_signal)
	{
		hipMemcpy(dSignalTL, h_signal, mMemSizeTL, hipMemcpyHostToDevice);
		hipfftExecC2C(planTL, dSignalTL, dSignalTL, HIPFFT_FORWARD);

		if (hipGetLastError() != hipSuccess) {
			fprintf(stderr, "FFT kernel launch failed: %s\n", hipGetErrorString(hipGetLastError()));
			return;
		}
		hipMemcpy(h_signal, dSignalTL, mMemSizeTL, hipMemcpyDeviceToHost);
	}
	/*void exeFFTNen(hipfftComplex *h_signal, hipfftComplex* h_image)
	{
	//move signal to gpu and process fft forward
	hipMemcpy(dSignalNen, h_signal, mMemSizeNen, hipMemcpyHostToDevice);
	hipfftExecC2C(planNenTH, dSignalNen, dSignalNen, HIPFFT_FORWARD);
	//move image to gpu and process fft forward
	hipMemcpy(dImageNen, h_image, mMemSizeNen, hipMemcpyHostToDevice);
	hipfftExecC2C(planNenTH, dImageNen, dImageNen, HIPFFT_FORWARD);
	// Element wise complext multiplication
	for (int i = 0; i < mFrameLen; i += 1024)
	{
	complexMulKernel << <1, 1024 >> >(dSignalNenRes+i, dSignalNen+i, dImageNen+i);
	}

	hipfftExecC2C(planNenTH, dSignalNenRes, dSignalNenRes, HIPFFT_FORWARD);

	hipMemcpy(h_signal, dSignalNenRes, mMemSizeNen, hipMemcpyDeviceToHost);
	}*/
	~coreFFT()
	{
		delete[] ramSignalTL;
		hipfftDestroy(planTL);
		//hipfftDestroy(planNenTH);
		// cleanup memory
		hipFree(dSignalTL);
		printf("\nmemory clear");
		//hipFree(dSignalNen);
		//hipFree(dImageNen);
	}
};
//_______________________________________________________________________

struct DataFrame// buffer for data frame
{
	char header[FRAME_HEADER_SIZE];
	char dataPM_I[FRAME_LEN];
	char dataPM_Q[FRAME_LEN];
	short dataLen;
	bool isToFFT;
} dataBuff[MAX_IREC];
//unsigned int gyroValue = 0;
udp_client_server::udp_client mUdp("127.0.0.1",31000);
u_char outputFrame[OUTPUT_FRAME_SIZE];

int iProcessing = 0, iReady = 50;
void packet_handler(u_char *param, const struct pcap_pkthdr *header, const u_char *pkt_data);
void pcapRun();

int mSocket;
void ProcessFrame(unsigned char*data, int len);

void *ProcessDataBuffer(void*);
void *ProcessCommandBuffer(void*);
pthread_t thread1,thread2;
void StartProcessing()
{
	pthread_create(&thread1,NULL,ProcessDataBuffer,NULL);
	pthread_create(&thread2,NULL,ProcessCommandBuffer,NULL);
}
coreFFT *mFFT;
FILE* pFile;
unsigned char buff[3000];


int main(int argc, char **argv)
{

	/* start the capture */
	mFFT = new coreFFT(FRAME_LEN, mFFTSize);

	StartProcessing();
	pcapRun();
	printf("\nNo interface available");
	getchar();

	return 0;
}

// Complex data type

void pcapRun()
{
	char error_buffer[PCAP_ERRBUF_SIZE];
	pcap_t *handle;
	int timeout_limit = 10000; /* In milliseconds */

	pcap_if_t *interfaces,*temp;
	int i=0;
	if(pcap_findalldevs(&interfaces,error_buffer)==-1)
	{
		printf("\nerror in pcap findall devs");
		return ;
	}

	printf("\n the interfaces present:");
	for(temp=interfaces;temp;temp=temp->next)
	{
		printf("\n%d  :  %s",i++,temp->name);

	}
	printf("\nOpen device to read:");
	if(pcap_findalldevs(&interfaces,error_buffer)==-1)
	{
		printf("\nerror in pcap findall devs");
		return ;
	}
	printf(" %s",interfaces->name);
	/* Open device for live capture */
	handle = pcap_open_live(
			interfaces->name,
			BUFSIZ,
			0,
			timeout_limit,
			error_buffer
		);
	if (handle == NULL) {
		 fprintf(stderr, "Could not open device: %s\n", error_buffer);
		 return ;
	 }

	pcap_loop(handle,NULL, packet_handler, NULL);
	    return ;
}
u_char dataOut[FRAME_LEN];
long int nFrames = 0;


char recvDatagram[1000];
void *ProcessCommandBuffer(void*)
{
	unsigned char watchDog[] = { 0xAA, 0xAA, 0xAA, 0xAA, 0xAA };
	while (true)
	{
		usleep(1000000);
		mUdp.send((char*)watchDog,4);
		//sendto(mSocket, (char*)watchDog, 4, 0, (struct sockaddr *) &si_peter, sizeof(si_peter));

		/*int PeterAddrSize = sizeof (si_peter);
		int iResult = recvfrom(mSocket, recvDatagram, 1000, 0, (struct sockaddr *) &si_peter, &PeterAddrSize);
		if (iResult == SOCKET_ERROR) {
			//wprintf(L"recvfrom failed with error %d\n", WSAGetLastError());
		}*/
	}

}
/*
int datatestI[MAX_IREC];
int datatestQ[MAX_IREC];
int datatestA[MAX_IREC];*/

void *ProcessDataBuffer(void*)
{
	while (true)
	{
		usleep(1000);
		while (iProcessing != iReady)
		{

			int dataLen = dataBuff[iProcessing].dataLen;
			for (int ir = 0; ir < dataLen; ir++)
			{

				//ramSignalNen[iProcessing][ir].x = sqrt(double(dataBuff[iProcessing].dataPM_I[ir] * dataBuff[iProcessing].dataPM_I[ir] + dataBuff[iProcessing].dataPM_Q[ir] * dataBuff[iProcessing].dataPM_Q[ir]));//int(dataBuff[iProcessing].dataPM_I[ir]);
				ramSignalNen[iProcessing][ir].x = float(dataBuff[iProcessing].dataPM_I[ir]);
				ramSignalNen[iProcessing][ir].y = float(dataBuff[iProcessing].dataPM_Q[ir]);//0;//
				//ramSignalNen[iProcessing][ir].y = 0;
			}
			if (!dataBuff[iProcessing].isToFFT || isPaused)
			{
				//jump to next period
				iProcessing++;
				if (iProcessing >= MAX_IREC)iProcessing = 0;
				continue;
			}
			for (int ir = 0; ir < dataLen; ir++)
			{
				int ia = iProcessing;
				for (int i = 0; i < mFFTSize; i++)
				{
					ramSignalTL[ir*mFFTSize + i] = ramSignalNen[ia][ir];
					ia--;
					if (ia < 0)ia += MAX_IREC;
				}
			}
			// perform fft
			if (mFFT->isActive)mFFT->exeFFTTL((hipfftComplex*)ramSignalTL);
			//dataBuff[iProcessing].header[32] = gyroValue >> 8;
			//dataBuff[iProcessing].header[33] = gyroValue;

			memcpy(outputFrame, dataBuff[iProcessing].header, FRAME_HEADER_SIZE);
			int fftSkip = BANG_KHONG*mFFTSize / 16;
			for (int i = 0; i < dataLen; i++)
			{
				double maxAmp = 0;
				int indexMaxFFT = 0;
				//for (int j = 0; j<FFT_SIZE_MAX; j++)

				for (int j = fftSkip; j < mFFTSize - fftSkip; j++)
				{
					double ampl = (ramSignalTL[i*mFFTSize + j].x * ramSignalTL[i*mFFTSize + j].x) + (ramSignalTL[i*mFFTSize + j].y * ramSignalTL[i*mFFTSize + j].y);
					if (ampl>maxAmp)
					{
						maxAmp = ampl;
						indexMaxFFT = j;
					}
				}
				double res = sqrt(double(maxAmp) / double(mFFTSize));
				if (res > 255)res = 255;
				outputFrame[i + FRAME_HEADER_SIZE] = u_char(res);// u_char(sqrt(float(maxAmp)) / float(FFT_SIZE_MAX));
				outputFrame[i + FRAME_LEN + FRAME_HEADER_SIZE] = u_char(indexMaxFFT*16.0 / (mFFTSize));
			}
			for (int i = dataLen; i < FRAME_LEN; i++)
			{
				outputFrame[i + FRAME_HEADER_SIZE] = 0;
				outputFrame[i + FRAME_LEN + FRAME_HEADER_SIZE] = 0;
			}
			mUdp.send( (char*)outputFrame, OUTPUT_FRAME_SIZE);
			//jump to next period
			iProcessing++;
			if (iProcessing >= MAX_IREC)iProcessing = 0;
		}



	}



}

#define UDP_HEADER_LEN 42
void packet_handler(u_char *param, const struct pcap_pkthdr *pkt_header, const u_char *pkt_data)
{
	//    struct tm ltime;
	//    char timestr[16];
	//    time_t local_tv_sec;

	/*
	* unused variables
	*/
	//    (VOID)(param);
	//    (VOID)(pkt_data);

	/* convert the timestamp to readable format */
	//    local_tv_sec = header->ts.tv_sec;
	//    localtime_s(&ltime, &local_tv_sec);
	//    strftime( timestr, sizeof timestr, "%H:%M:%S", &ltime);
	if (pkt_header->len<1000)return;
	//int port = ((*(pkt_data + 36) << 8) | (*(pkt_data + 37)));

	if (
		((*(pkt_data + 6)) == 0) &&
		((*(pkt_data + 7)) == 0x12) &&
		((*(pkt_data + 8)) == 0x34)

		)
	{
		/*
		+ 0: 1024 byte đầu kênh I
		+ 1: 1024 byte sau kênh I
		+ 2: 1024 byte đầu kênh Q
		+ 3: 1024 byte sau kênh Q
		+ 4: 256 byte máy hỏi
		+ 5: 1024 byte tín hiệu giả L/tục (512 byte đầu là I, 512 byte sau là Q)
		+ 6: 1024 byte sau kênh I tín hiệu xung đơn
		+ 7: 1024 byte sau kênh Q tín hiệu xung đơn

		*/
		u_char* data = (u_char*)pkt_data + UDP_HEADER_LEN;
		ProcessFrame(data, pkt_header->len);


	}

}
/*void packet_handler_compress(u_char *param, const struct pcap_pkthdr *pkt_header, const u_char *pkt_data)
{
if (pkt_header->len<1000)return;
if (((*(pkt_data + 36) << 8) | (*(pkt_data + 37))) != 5000)
{
return;
}
u_char* data = (u_char*)pkt_data + UDP_HEADER_LEN;
if (data[0] == 0)		//I chanel first part
{
iReady++;
if (iReady >= MAX_IREC)iReady = 0;
memcpy(dataBuff[iReady].header, data, FRAME_HEADER_SIZE);
memcpy(dataBuff[iReady].dataPM_I, data + FRAME_HEADER_SIZE, 1024);
}
else if (data[0] == 2) //Q chanel first part
{
memcpy(dataBuff[iReady].dataPM_Q, data + FRAME_HEADER_SIZE, 1024);
}
else if (data[0] == 1) //I chanel second part
{
memcpy(dataBuff[iReady].dataPM_I + 1024, data + FRAME_HEADER_SIZE, 1024);
}
else if (data[0] == 3) //Q chanel second part
{
memcpy(dataBuff[iReady].dataPM_Q + 1024, data + FRAME_HEADER_SIZE, 1024);
}
return;
}*/
/*
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   STT |   Byte    |   Chức                                              |
|       |           |   năng                                              |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   1   |   0       |   Id gói                                            |
|       |           |   tin:                                              |
|       |           |   0,1,2,3:                                          |
|       |           |   iq th mã pha (mỗi kênh 2048 byte)                 |
|       |           |   4: 256                                            |
|       |           |   byte máy hỏi, mỗi bít một o_cu_ly                 |
|       |           |   5: iq th                                          |
|       |           |   giả liên tục, 512 byte i, 512 byte q              |
|       |           |   6,7: iq                                           |
|       |           |   cho tín hiệu xung đơn, mỗi kênh 1024 byte         |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   2   |   1, 2, 3 |   Byte cho                                          |
|       |           |   báo hỏng:                                         |
|       |           |   1: loại                                           |
|       |           |   mô-đun, (0, 1, 2, 3)                              |
|       |           |   2: Loại                                           |
|       |           |   tham số (bb, cc, dd)                              |
|       |           |   3: Tham                                           |
|       |           |   số mô-đun                                         |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   3   |   4       |   Phân giải                                         |
|       |           |   ra đa: 0 (15m), 1 (30m)......                     |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   4   |   5,6     |   Loại tín                                          |
|       |           |   hiệu phát và tham số:                             |
|       |           |   5: loại                                           |
|       |           |   th phát (0: xung đơn; 1: mã pha; 2: giả ltuc)     |
|       |           |   6: tham                                           |
|       |           |   số cho loại th trên                               |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   5   |   7,8     |   Hai byte                                          |
|       |           |   trung bình tạp máy thu (ktra báo hỏng tuyến thu)  |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   6   |   9, 10,  |   4 byte                                            |
|       |   11, 12  |   quay an-ten                                       |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   7   |   13, 14  |   Hai byte                                          |
|       |           |   hướng tàu                                         |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   8   |   15, 16  |   Hai byte                                          |
|       |           |   hướng mũi tàu                                     |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   9   |   17, 18  |   Hai byte                                          |
|       |           |   tốc độ tàu                                        |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   10  |   19      |   Thông                                             |
|       |           |   báo chế độ chủ đông - bị động, tốc độ quay an-ten |
|       |           |   - bít thấp                                        |
|       |           |   thông báo cđ-bđ (1: chủ động)                     |
|       |           |   - 4 bít                                           |
|       |           |   cao là tốc độ an-ten                              |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   11  |   20      |   Thông                                             |
|       |           |   báo tần số phát và đặt mức tín hiệu:              |
|       |           |   - 4 bít                                           |
|       |           |   cuối là tần số phát                               |
|       |           |   - 4 bít                                           |
|       |           |   cao là đặt mức th                                 |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   12  |   21      |   Thông                                             |
|       |           |   báo chọn thang cự ly và bật/tắt AM2:              |
|       |           |   - 4 bít                                           |
|       |           |   cuối là thang cự ly (0: 2 lý; 1: 4 lý.....)       |
|       |           |   - 4 bít                                           |
|       |           |   cao là báo bật/tắt AM2: 0: tắt, 1: bật            |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   13  |   22      |   Thông                                             |
|       |           |   báo số điểm FFT:                                  |
|       |           |   1(fft8);                                          |
|       |           |   2(fft16) ;...;32(fft256)                          |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
Id gói                                            |
|       |           |   tin:                                              |
|       |           |   0,1,2,3:                                          |
|       |           |   iq th mã pha (mỗi kênh 2048 byte)                 |
|       |           |   4: 256                                            |
|       |           |   byte máy hỏi, mỗi bít một o_cu_ly                 |
|       |           |   5: iq th                                          |
|       |           |   giả liên tục, 512 byte i, 512 byte q              |
|       |           |   6,7: iq                                           |
|       |           |   cho tín hiệu xung đơn, mỗi kênh 1024 byte
*/
static int fftID = -1;
void ProcessFrame(unsigned char*data, int len)
{
	int iNext = iReady + 1;
	if (iNext >= MAX_IREC)iNext = 0;
	int newfftID = data[22];
	if(fftID!=newfftID)
	{
		if (newfftID > 8 || newfftID < 0)
		{
			printf("\nWrong fftID");
			return;
		}
		fftID = newfftID;
		mFFTSize = pow(2.0, fftID + 2);
		if (mFFTSize > 512 || mFFTSize < 4)mFFTSize = 32;
		isPaused = true;

		usleep(100000);
		iProcessing = iReady;
		if (mFFT)
		{
			delete mFFT;
		}
		mFFT = new coreFFT(FRAME_LEN, mFFTSize);

		usleep(50000);
		isPaused = false;
	}
	memcpy(dataBuff[iNext].header, data, FRAME_HEADER_SIZE);

	//printf("data[0]=%d\n", data[0]);
	bool isLastFrame = false;
	if (data[0] == 0)		//0: 1024 byte đầu kênh I
	{
		//memcpy(dataBuff[iNext].header, data, FRAME_HEADER_SIZE);
		memcpy(dataBuff[iNext].dataPM_I, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 1) //1: 1024 byte sau kênh I
	{
		memcpy(dataBuff[iNext].dataPM_I + 1024, data + FRAME_HEADER_SIZE, 1024);

	}
	else if (data[0] == 2) //2: 1024 byte đầu kênh Q
	{
		memcpy(dataBuff[iNext].dataPM_Q, data + FRAME_HEADER_SIZE, 1024);

	}
	else if (data[0] == 3) //3: 1024 byte sau kênh Q
	{
		memcpy(dataBuff[iNext].dataPM_Q + 1024, data + FRAME_HEADER_SIZE, 1024);
		dataBuff[iNext].dataLen = FRAME_LEN;
		isLastFrame = true;

	}
	else if (data[0] == 4) //4: máy hỏi
	{

		mUdp.send( (char*)data, len);

		//isLastFrame = true;

	}
	else if (data[0] == 5) //5: 1024 byte tín hiệu giả L/tục (512 byte đầu là I, 512 byte sau là Q)
	{
		memcpy(dataBuff[iNext].dataPM_I, data + FRAME_HEADER_SIZE, 512);
		memcpy(dataBuff[iNext].dataPM_Q, data + FRAME_HEADER_SIZE + 512, 512);
		dataBuff[iNext].dataLen = 512;
		isLastFrame = true;
	}
	else if (data[0] == 6) //6: 1024 byte kênh I tín hiệu xung đơn
	{
		memcpy(dataBuff[iNext].dataPM_I, data + FRAME_HEADER_SIZE, 1024);
		dataBuff[iNext].dataLen = 1024;

	}
	else if (data[0] == 7) //7: 1024 byte kênh Q tín hiệu xung đơn
	{
		memcpy(dataBuff[iNext].dataPM_Q, data + FRAME_HEADER_SIZE, 1024);
		dataBuff[iNext].dataLen = 1024;
		isLastFrame = true;

	}
	if (isLastFrame)
	{
		iReady++;
		dataBuff[iNext].isToFFT = ((iNext%FFT_STEP) == 0);
		if (iReady >= MAX_IREC)iReady = 0;
	}
	return;

}
